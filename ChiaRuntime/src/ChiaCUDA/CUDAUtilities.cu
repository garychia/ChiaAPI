#include "ChiaCUDA/CUDAUtilities.cuh"

#include <iostream>

bool CheckCUDAStatus(const hipError_t &error, const char *fileName, unsigned int line)
{
    return error == hipSuccess;
}

void PrintCUDAErrorMessage(const hipError_t &error, bool newline)
{
    if (error == hipSuccess)
        return;
    std::printf("%s", hipGetErrorString(error));
    if (newline)
        std::printf("\n");
}
